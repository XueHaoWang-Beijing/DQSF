
#include <hip/hip_runtime.h>
__global__ void addVectorsMask(float *A, float *B, float *C, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i!= size)
        return;

    C[i] = A[i] + B[i];
}

void addVectors(float *A, float *B, float *C, int size)
{
    float *devPtrA = 0,*devPtrB = 0,*devPtrC = 0;
    hipMalloc(&devPtrA,sizeof(float)* size);
    hipMalloc(&devPtrB,sizeof(float)* size);
    hipMalloc(&devPtrC,sizeof(float)* size);

    hipMemcpy(devPtrA,A, sizeof(float)* size, hipMemcpyHostToDevice);
    hipMemcpy(devPtrB,B, sizeof(float)* size, hipMemcpyHostToDevice);
    addVectorsMask<<<int(size/1024),1024>>>(devPtrA,devPtrB, devPtrC, size);
    hipMemcpy(C,devPtrC, sizeof(float)* size, hipMemcpyDeviceToHost);

    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);

}